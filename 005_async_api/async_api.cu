#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#define ADDITION_CONST 50

__global__ void kernel_addition(int *a){
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	a[idx] += ADDITION_CONST;
}

bool verify_result(int *a, int length){
	int i = 0;
	for(i = 0; i < length; i++){
		if(a[i] != ADDITION_CONST)
			return false;
	}
	return true;
}

int main(int argc, char **argv){

	int deviceId;
	hipDeviceProp_t deviceProps;

	deviceId = findCudaDevice(argc, (const char**)argv);
	printf("Cuda device found: %d\n", deviceId);

	checkCudaErrors(hipGetDeviceProperties(&deviceProps, deviceId));
	printf("Cuda Device Name: %s\n", deviceProps.name);

	int n = 16 * 1024 * 1024;
	int nbytes = n * sizeof(int);
	int blocksize = 256;
	int threads = 512;

	int *a = NULL;
	int *d_a = NULL;

	checkCudaErrors(hipHostMalloc((void**)&a, nbytes));
	memset(a, 0, nbytes);

	checkCudaErrors(hipMalloc((void**)&d_a, nbytes));
	checkCudaErrors(hipMemset(d_a, 0, nbytes));

	hipEvent_t start, stop;

	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));

	checkCudaErrors(hipEventRecord(start, 0));
	checkCudaErrors(hipMemcpyAsync(d_a, a, nbytes, hipMemcpyHostToDevice, 0));
	kernel_addition<<< n/blocksize, threads, 0 , 0>>>(d_a);
	checkCudaErrors(hipMemcpyAsync(a, d_a, nbytes, hipMemcpyDeviceToHost, 0));
	checkCudaErrors(hipEventRecord(stop, 0));
	checkCudaErrors(hipEventSynchronize(stop));

	float gpu_time = 0.0f;
	checkCudaErrors(hipEventElapsedTime(&gpu_time, start, stop));

	printf("Result validation: %d\n", verify_result(a, n));
	printf("GPU Time: %f ms\n", gpu_time);

	checkCudaErrors(hipEventDestroy(start));
	checkCudaErrors(hipEventDestroy(stop));
	checkCudaErrors(hipHostFree(a));
	checkCudaErrors(hipFree(d_a));

	return 0;
}
