#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
#include <helper_functions.h>


__global__ void sin_cos_kernel(float *a, int offset){
	int idx = offset + threadIdx.x + blockIdx.x * blockDim.x;
	a[idx] = sinf((float)(idx)) + cosf((float)(idx));
}


bool verify_data(float *src, float *dst, int length){
	printf("Verifying: %d elements\n", length);
	for(int i = 0; i < length - 1; i++){
		if(src[i] != dst[i]){
			printf("Result failed at: %d\n", i);
			return false;
		}
	}
	return true;
}

int main(int argc, char **argv){
	int deviceId;
	hipDeviceProp_t deviceProps;
	int nstreams = 8;
	int blocksize = 256;
	int n = 16 * 1024 * blocksize * nstreams;
	int streamsize = n / nstreams;
	int streambytes = streamsize * sizeof(float);
	int nbytes = n * sizeof(float);

	deviceId = findCudaDevice(argc, (const char**)argv);
	printf("Cuda device found: %d\n", deviceId);

	checkCudaErrors(hipGetDeviceProperties(&deviceProps, deviceId));
	printf("Cuda device name: %s\n", deviceProps.name);

	if(!deviceProps.concurrentKernels){
		printf("Concurrent kernels are not supported\n");
		return -1;
	}
	printf("Concurrent kernels are supported: number of concurrent kernels: %d\n", deviceProps.multiProcessorCount);
	printf("============================================\n\n");

	
	float *a = NULL;
	checkCudaErrors(hipHostMalloc((void**)&a, nbytes));
	memset(a, 0, nbytes);

	float *verify = NULL;
	checkCudaErrors(hipHostMalloc((void**)&verify, nbytes));
	memset(verify, 0, nbytes);

	float *d_a = NULL;
	checkCudaErrors(hipMalloc((void**)&d_a, nbytes));
	checkCudaErrors(hipMemset(a, 0, nbytes));

	hipEvent_t start, stop;
	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));

	// serial execution of the entire data
	hipEventRecord(start, 0);
	checkCudaErrors(hipMemcpy(d_a, a, nbytes, hipMemcpyHostToDevice));
	sin_cos_kernel<<<n/blocksize, blocksize>>>(d_a, 0);
	checkCudaErrors(hipMemcpy(a, d_a, nbytes, hipMemcpyDeviceToHost));
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	// copy the result to the verify variable, 
	// so that we can verify the concurrent kernel results
	memcpy(verify, a, nbytes);

	printf("Verifying result: %d\n", verify_data(a, verify, n));

	float gpu_time = 0.0f;
	checkCudaErrors(hipEventElapsedTime(&gpu_time, start, stop));
	printf("Time taken for serial execution: %f ms\n", gpu_time);
	printf("============================================\n\n");


	hipStream_t streams[nstreams];
	for(int i = 0; i < nstreams; i++){
		checkCudaErrors(hipStreamCreate(&streams[i]));
	}

	// running the streams in loop of HDCopy, kernel, DHCopy
	memset(a, 0, nbytes);
	checkCudaErrors(hipEventRecord(start, 0));
	for(int i = 0; i < nstreams; i++){
		int offset = i * streamsize;
		checkCudaErrors(hipMemcpyAsync(&d_a[offset], &a[offset], streambytes, hipMemcpyHostToDevice, streams[i]));
		sin_cos_kernel<<<streamsize/blocksize, blocksize, 0, streams[i]>>>(d_a, offset);
		checkCudaErrors(hipMemcpyAsync(&a[offset], &d_a[offset], streambytes, hipMemcpyDeviceToHost, streams[i]));
	}
	checkCudaErrors(hipEventRecord(stop, 0));
	checkCudaErrors(hipEventSynchronize(stop));

	gpu_time = 0.0f;
	checkCudaErrors(hipEventElapsedTime(&gpu_time, start, stop));
	printf("%d stream execution time: %f ms\n", nstreams, gpu_time);
	printf("Stream result verification: %d\n", verify_data(a, verify, n));
	printf("=============================================\n\n");


	// running the streams in individual loops - loop HDCopy, loop kernel, loop DHCopy
	memset(a, 0, nbytes);
	checkCudaErrors(hipEventRecord(start, 0));
	for(int i = 0; i < nstreams; i++){
		int offset = i * streamsize;
		checkCudaErrors(hipMemcpyAsync(&d_a[offset], &a[offset], streambytes, hipMemcpyHostToDevice, streams[i]));
	}
	for(int i = 0; i < nstreams; i++){
		int offset = i * streamsize;
		sin_cos_kernel<<<streamsize/blocksize, blocksize, 0, streams[i]>>>(d_a, offset);
	}
	for(int i = 0; i < nstreams; i++){
		int offset = i * streamsize;
		checkCudaErrors(hipMemcpyAsync(&a[offset], &d_a[offset], streambytes, hipMemcpyDeviceToHost, streams[i]));
	}
	checkCudaErrors(hipEventRecord(stop, 0));
	checkCudaErrors(hipEventSynchronize(stop));
	gpu_time = 0.0f;
	checkCudaErrors(hipEventElapsedTime(&gpu_time, start, stop));
	printf("%d stream execution time individual loop: %f ms\n", nstreams, gpu_time);
	printf("Stream result verification: %d\n", verify_data(a, verify, n));

	for(int i = 0; i < nstreams; i++){
		checkCudaErrors(hipStreamDestroy(streams[i]));
	}

	checkCudaErrors(hipHostFree(a));
	checkCudaErrors(hipHostFree(verify));
	checkCudaErrors(hipFree(d_a));

	return 0;
}
