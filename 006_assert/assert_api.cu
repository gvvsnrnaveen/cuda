#include "hip/hip_runtime.h"
#include <stdio.h>
#include <unistd.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

__global__ void testKernel(int N){
	int gid = blockIdx.x * blockDim.x + threadIdx.x;
	assert(gid < N);
}

int main(int argc, char **argv){
	int deviceId; 
	hipDeviceProp_t deviceProp;
	hipError_t error;

	int blocks = 2;
	int threads = 32;

	deviceId = findCudaDevice(argc, (const char**)argv);
	printf("Device id found: %d\n", deviceId);

	checkCudaErrors(hipGetDeviceProperties(&deviceProp, deviceId));
	printf("Device Name: %s\n", deviceProp.name);

	dim3 dimGrid(blocks);
	dim3 dimBlock(threads);

	testKernel<<<dimGrid, dimBlock>>>(60);

	printf("Begin: Assert\n");
	error = hipDeviceSynchronize();
	printf("End: Assert\n");

	if(error == hipErrorAssert){
		printf("Cuda Assert: %s\n", hipGetErrorString(error));
	}

	return 0;
}
