
#include <hip/hip_runtime.h>
extern "C" __global__ void kernel_vector_addition(const float *a, const float *b, float *c, int n){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n) {
		c[i] = a[i] + b[i];
	}
}
