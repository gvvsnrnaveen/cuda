#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

__global__ void vector_increment_kernel(int *ga, int value){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	ga[idx] += value;
}

bool check_output(int *a, int n, int value){
	for(int i = 0; i < n; i++){
		if( a[i] != value ){
			return false;
		}
	}
	return true;
}

int main(int argc, char **argv){
	int numberOfDevices = 0;
	int deviceId;
	hipDeviceProp_t deviceProps;

	hipGetDeviceCount(&numberOfDevices);
	printf("Number of devices: %d\n", numberOfDevices);

	deviceId = findCudaDevice(argc, (const char**)argv);
	checkCudaErrors(hipGetDeviceProperties(&deviceProps, deviceId));
	printf("Cuda Device Name: %s\n", deviceProps.name);

	int n = 16 * 1024 * 1024;
	int nbytes = n * sizeof(int);
	int value = 26;

	size_t gpu_free_mem, gpu_total_mem;
	checkCudaErrors(hipMemGetInfo(&gpu_free_mem, &gpu_total_mem));
	printf("GPU Mem stats: free - %ld, total - %ld\n", gpu_free_mem, gpu_total_mem);

	int *a, *d_a;
	checkCudaErrors(hipHostMalloc((void**)&a, nbytes));
	memset(a, 0, nbytes);

	checkCudaErrors(hipMalloc((void**)&d_a, nbytes));
	checkCudaErrors(hipMemset(a, 0, nbytes));

	checkCudaErrors(hipMemGetInfo(&gpu_free_mem, &gpu_total_mem));
	printf("GPU Mem stats: free - %ld, total - %ld\n", gpu_free_mem, gpu_total_mem);

	dim3 threads = dim3(512, 1);
	dim3 blocks = dim3( n /threads.x, 1);


	hipMemcpy(d_a, a, nbytes, hipMemcpyHostToDevice);

	vector_increment_kernel<<<blocks, threads, 0, 0>>>(d_a, value);
	hipMemcpy(a, d_a, nbytes, hipMemcpyDeviceToHost);

	printf("Result: %d\n", check_output(a, n, value));

	checkCudaErrors(hipHostFree(a));
	checkCudaErrors(hipFree(d_a));

	checkCudaErrors(hipMemGetInfo(&gpu_free_mem, &gpu_total_mem));
	printf("GPU Mem stats: free - %ld, total - %ld\n", gpu_free_mem, gpu_total_mem);

	return 0;
}
