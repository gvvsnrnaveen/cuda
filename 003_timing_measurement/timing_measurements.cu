#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

__global__ void addition_kernel(int *a, int *b, int *c){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	c[idx] = a[idx] + b[idx];
}

__global__ void subtraction_kernel(int *a, int *b, int *c){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	c[idx] = b[idx] - a[idx];
}

bool validate_addition_result(int *c, int n){
	int i = 0;
	for(i = 0; i < n; i++){
		if(c[i] != ((3 * i) + (5 * i)))
			return false;
	}
	return true;
}

bool validate_subtraction_result(int *d, int n){
	int i = 0;
	for(i = 0; i < n; i++){
		if(d[i] != ((5 * i) - (3 * i)))
			return false;
	}
	return true;
}

int main(int argc, char **argv){
	int devId;
	int numberOfDevices = 0;
	hipDeviceProp_t deviceProps;

	hipGetDeviceCount(&numberOfDevices);
	printf("Number of cuda devices: %d\n", numberOfDevices);

	devId = findCudaDevice(argc, (const char**)argv);
	printf("Device ID found: %d\n", devId);

	checkCudaErrors(hipGetDeviceProperties(&deviceProps, devId));
	printf("Device Name: %s\n", deviceProps.name);

	int n = 16 * 1024 * 1024;
	int nbytes = n * sizeof(int);
	int i = 0;
	int *a = NULL, *b = NULL, *c = NULL, *d = NULL;
	checkCudaErrors(hipHostMalloc((void**)&a, nbytes));
	memset(a, 0, nbytes);
	checkCudaErrors(hipHostMalloc((void**)&b, nbytes));
	memset(b, 0, nbytes);
	checkCudaErrors(hipHostMalloc((void**)&c, nbytes));
	memset(c, 0, nbytes);
	checkCudaErrors(hipHostMalloc((void**)&d, nbytes));
	memset(d, 0, nbytes);

	for(i=0; i < n; i++){
		a[i] = 3 * i;
		b[i] = 5 * i;
	}

	int *d_a = NULL, *d_b = NULL, *d_c = NULL, *d_d = NULL;
	checkCudaErrors(hipMalloc((void**)&d_a, nbytes));
	checkCudaErrors(hipMemset(d_a, 0, nbytes));
	checkCudaErrors(hipMalloc((void**)&d_b, nbytes));
	checkCudaErrors(hipMemset(d_b, 0, nbytes));
	checkCudaErrors(hipMalloc((void**)&d_c, nbytes));
	checkCudaErrors(hipMemset(d_c, 0, nbytes));
	checkCudaErrors(hipMalloc((void**)&d_d, nbytes));
	checkCudaErrors(hipMemset(d_d, 0, nbytes));

	dim3 threads = dim3(512, 1);
	dim3 blocks = dim3( n / threads.x, 1);

	hipEvent_t start, stop;

	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));

	checkCudaErrors(hipDeviceSynchronize());
	checkCudaErrors(hipProfilerStart());

	hipMemcpy(d_a, a, nbytes, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, nbytes, hipMemcpyHostToDevice);
	hipEventRecord(start, 0);
	
	addition_kernel<<<blocks, threads, 0, 0>>>(d_a, d_b, d_c);
	subtraction_kernel<<<blocks, threads, 0, 0>>>(d_a, d_b, d_d);
	hipEventRecord(stop, 0);
	hipMemcpy(c, d_c, nbytes, hipMemcpyDeviceToHost);
	hipMemcpy(d, d_d, nbytes, hipMemcpyDeviceToHost);
	hipEventSynchronize(stop);

	float gpu_time = 0.0f;
	checkCudaErrors(hipEventElapsedTime(&gpu_time, start, stop));

	printf("GPU Time: %.2fms\n", gpu_time);
	printf("Addition Result is: %d\n", validate_addition_result(c, n));
	printf("Subtraction Result is: %d\n", validate_subtraction_result(d, n));
	
	checkCudaErrors(hipHostFree(a));
	checkCudaErrors(hipHostFree(b));
	checkCudaErrors(hipHostFree(c));
	checkCudaErrors(hipFree(d_a));
	checkCudaErrors(hipFree(d_b));
	checkCudaErrors(hipFree(d_c));
	return 0;
}
